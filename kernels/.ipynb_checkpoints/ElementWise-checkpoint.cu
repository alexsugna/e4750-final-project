
#include <hip/hip_runtime.h>
// element-wise matrix operations

__global__ void MatEleMul(float* A, float* B, float *C, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        C[index] = A[index] * B[index];
    }
}

__global__ void MatEleAdd(float* A, float* B, float *C, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        C[index] = A[index] + B[index];
    }
}

__global__ void MatEleSubtract(float* A, float* B, float *C, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        C[index] = A[index] - B[index];
    }
}

__global__ void MatEleSquare(float* A, float *C, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        C[index] = A[index] * A[index];
    }
}

__global__ void MatEleDivide(float* A, float* B, float *C, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        C[index] = A[index] / B[index];
    }
}

__global__ void MatEleAddInPlace(float* A, float B, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        A[index] = A[index] + B;
    }
}

__global__ void MatEleMulInPlace(float* A, float* B, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        A[index] = A[index] * B[index];
    }
}

__global__ void MatEleDivideInPlace(float* A, float* B, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        A[index] = A[index] / B[index];
    }
}

__global__ void MatEleSubtractInPlace(float* A, float* B, int height, int width){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(x < width && y < height){
        int index = y * width + x;
        A[index] = A[index] - B[index];
    }
}
